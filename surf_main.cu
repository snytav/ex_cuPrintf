#include "hip/hip_runtime.h"
///// https://forums.developer.nvidia.com/t/can-i-use-operator-new-in-device-code/37594ii

#include "surf2Dread.h"
#include<stdio.h>
#include "cuPrintf.cu"

SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	*d_s = new SurfaceEmulator(m,n);
    (*d_s)->write(2,3,3.14);
    double d = 0.0;
    d = (*d_s)->read(2,3);
    printf("surf read %e\n",d);

}


int main(){


  cudaPrintfInit();
  kernel<<<1,1>>>(&d_surf,3,5);
  hipDeviceSynchronize();
  cudaPrintfDisplay();


}
