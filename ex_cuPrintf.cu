#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

#include "stdio.h"

#include "surf_emulator.h"

__global__ void device_greetings(void){

        cuPrintf("Hello world, form the device %d \n",5);

}


SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	*d_s = new SurfaceEmulator(m,n);
    (*d_s)->write(2,3,3.14);
    double d = 0.0;
    d = (*d_s)->read(2,3);
    printf("surf read %e\n",d);

}




int main(){

        //greet from the host

        printf("Helo world, from the host\n");

//intitialize cuPrintf

        cudaPrintfInit();

//launch the kernel with a single thread

        device_greetings<<<1,1>>>();

//display the device's greetings

        cudaPrintfDisplay();

//clean up

        cudaPrintfEnd();

return 0;

}
