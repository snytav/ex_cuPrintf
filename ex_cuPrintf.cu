#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

#include "stdio.h"

__global__ void device_greetings(void){

        cuPrintf("Hello world, form the device %d \n",5);

}

int main(){

        //greet from the host

        printf("Helo world, from the host\n");

//intitialize cuPrintf

        cudaPrintfInit();

//launch the kernel with a single thread

        device_greetings<<<1,1>>>();

//display the device's greetings

        cudaPrintfDisplay();

//clean up

        cudaPrintfEnd();

return 0;

}
