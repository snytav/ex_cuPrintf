#include "hip/hip_runtime.h"
#include "cuPrintf.cu"

#include "stdio.h"

#include "surf_emulator.h"

__global__ void device_greetings(void){

        cuPrintf("Hello world, form the device %d \n",5);

}


SurfaceEmulator *d_surf;

__global__ void kernel(SurfaceEmulator **d_s,int n,int m){
	//*d_s = new SurfaceEmulator(m,n);
 //   (*d_s)->write(2,3,3.14);
//    double d = 0.0;
//    d = (*d_s)->read(2,3);
    cuPrintf("surf read \n");

}




int main(){

        //greet from the host

        //printf("Hello world, from the host\n");

//intitialize cuPrintf

        cudaPrintfInit();

//launch the kernel with a single thread

        kernel<<<1,1>>>(&d_surf,3,5);

//display the device's greetings

        cudaPrintfDisplay();

//clean up

        cudaPrintfEnd();

return 0;

}
